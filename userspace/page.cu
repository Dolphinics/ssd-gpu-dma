#include <hip/hip_runtime.h>
#include "page.h"
#ifdef __cplusplus
extern "C" {
#endif
#include <stddef.h>
#include <stdint.h>
#include <stdlib.h>
#include <sys/mman.h>
#include <unistd.h>
#include <errno.h>
#include <cunvme_ioctl.h>
#include <sys/ioctl.h>
#include <fcntl.h>
#include <stdio.h>
#include <string.h>



static int pin_ram_page(page_t* page, int fd)
{
    struct cunvme_pin_page request;
    request.handle = CUNVME_NO_HANDLE;
    request.paddr = (uint64_t) NULL;
    request.vaddr = (uint64_t) page->virt_addr;

    if (ioctl(fd, CUNVME_PIN, &request) < 0)
    {
        fprintf(stderr, "ioctl to kernel failed: %s\n", strerror(errno));
        return errno;
    }

    page->kernel_handle = request.handle;
    page->phys_addr = request.paddr;
    return 0;
}


static int get_gpu_page(page_t* page, size_t size, int fd, int dev)
{
    // TODO: Copy magic from rdma bench
    return 0;
}


int get_page(page_t* page, int fd, int dev)
{
    int err;

    long page_size = sysconf(_SC_PAGESIZE);
    if (page_size == -1)
    {
        fprintf(stderr, "Failed to retrieve page size: %s\n", strerror(errno));
        return errno;
    }
    
    if (dev >= 0)
    {
        return get_gpu_page(page, page_size, fd, dev);
    }

    void* addr = mmap(NULL, page_size, PROT_READ | PROT_WRITE, MAP_ANONYMOUS | MAP_PRIVATE, -1, 0);
    if (addr == NULL)
    {
        fprintf(stderr, "Failed to mmap page: %s\n", strerror(errno));
        return errno;
    }

    if (mlock(addr, page_size) != 0)
    {
        fprintf(stderr, "Failed to mlock page: %s\n", strerror(errno));
        err = errno;
        munmap(addr, page_size);
        return err;
    }

    page->device = -1;
    page->kernel_handle = CUNVME_NO_HANDLE;
    page->virt_addr = addr;
    page->phys_addr = (uint64_t) NULL;
    page->page_size = page_size;

    if (pin_ram_page(page, dev) != 0)
    {
        put_page(page, -1);
        return EIO;
    }

    return 0;
}


static void put_gpu_page(page_t* page, int fd)
{
}


void put_page(page_t* page, int fd)
{
    if (page->device >= 0)
    {
        put_gpu_page(page, fd);
        return;
    }

    if (fd > 0)
    {
    }

    munlock(page->virt_addr, page->page_size);
    munmap(page->virt_addr, page->page_size);
}



#ifdef __cplusplus
}
#endif
