#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "memory.h"
#include "nvm/types.h"
#include "nvm/queue.h"
#include "nvm/command.h"
#include "nvm/util.h"
#include "nvm/ctrl.h"
#include <cstdio>
#include <cstddef>
#include <cstring>
#include <sys/types.h>
#include <unistd.h>
#include <errno.h>

//
//__host__ __device__
//static int prepare_write_cmd(nvm_queue_t sq, uint32_t ns_id, uint32_t blk_size, memory_t* buffer, uint64_t start_lba, uint16_t n_blks)
//{
//    struct command* cmd = sq_enqueue(sq);
//    if (cmd == NULL)
//    {
//        return EAGAIN;
//    }
//
//    cmd_header(cmd, NVM_WRITE, ns_id);
//    cmd_data_ptr(cmd, NULL, buffer, 1);
//    cmd->dword[10] = (uint32_t) start_lba;
//    cmd->dword[11] = (uint32_t) (start_lba >> 1);
//
//    cmd->dword[12] = n_blks;
//    cmd->dword[13] = 0;
//    cmd->dword[14] = 0;
//    cmd->dword[15] = 0;
//    return 0;
//}
//
//__host__ __device__
//static int prepare_read_cmd(nvm_queue_t sq, uint32_t ns_id, uint32_t blk_size, memory_t* buffer, uint64_t start_lba, uint16_t n_blks)
//{
//    struct command* cmd = sq_enqueue(sq);
//    if (cmd == NULL)
//    {
//        return EAGAIN;
//    }
//
//    //size_t len = (n_blks * blk_size) / buffer->page_size;
//
//    cmd_header(cmd, NVM_READ, ns_id);
//    //cmd_data_ptr(cmd, NULL, buffer, _MIN(buffer->n_addrs, len));
//    cmd_data_ptr(cmd, NULL, buffer, 1);
//
//    cmd->dword[10] = (uint32_t) start_lba;
//    cmd->dword[11] = (uint32_t) (start_lba >> 32);
//
//    cmd->dword[12] = (0x00 << 31) | (0x00 << 30) | (0x00 << 26) | n_blks;
//
//    cmd->dword[13] = 0;
//    cmd->dword[14] = 0;
//    cmd->dword[15] = 0;
//
//    return 0;
//}
//
//
//__global__ void do_work(memory_t* buffer, nvm_queue_t sq, uint32_t* tailst) //, nvm_queue_t cq)
//{
//    *tailst = 0;
//    if (prepare_read_cmd(sq, 1, 512, buffer, 0, 1) == 0)
//    {
//        *tailst = sq->tail;
//        sq_submit(sq); // this works
//    }
//
//    //while (cq_poll(cq) == NULL);
//
//    //*result = *((uint32_t*) buffer->virt_addr);
//}
//
//
//static int create_queues(int ioctl_fd, nvm_controller_t ctrl, int dev, nvm_queue_t* cq, nvm_queue_t* sq)
//{
//    int err;
//
//    err = nvm_prepare_queues(ctrl, cq, sq);
//    if (err != 0)
//    {
//        fprintf(stderr, "Failed to prepare queue handles\n");
//        return err;
//    }
//
//    err = get_gpu_page(ioctl_fd, dev, &((*cq)->page));
//    //err = get_ram_page(ioctl_fd, &((*cq)->page));
//    if (err != 0)
//    {
//        fprintf(stderr, "Failed to allocate queue memory\n");
//        return ENOMEM;
//    }
//    //memset((*cq)->page.virt_addr, 0, (*cq)->page.page_size);
//    hipMemset((*cq)->page.virt_addr, 0, (*cq)->page.page_size);
//
//    err = get_gpu_page(ioctl_fd, dev, &((*sq)->page));
//    //err = get_ram_page(ioctl_fd, &((*sq)->page));
//    if (err != 0)
//    {
//        fprintf(stderr, "Failed to allocate queue memory\n");
//        return ENOMEM;
//    }
//    //memset((*sq)->page.virt_addr, 0, (*sq)->page.page_size);
//    hipMemset((*sq)->page.virt_addr, 0, (*sq)->page.page_size);
//
//    err = nvm_commit_queues(ctrl);
//    if (err != 0)
//    {
//        fprintf(stderr, "Failed to commit prepared queues\n");
//        return err;
//    }
//
//    return 0;
//}
//
//
//extern "C" __host__
//int cuda_workload(int ioctl_fd, nvm_controller_t ctrl, int dev, void* reg_ptr, size_t reg_len)
//{
//    hipError_t err = hipSetDevice(dev);
//    if (err != hipSuccess)
//    {
//        fprintf(stderr, "Failed to set CUDA device: %s\n", hipGetErrorString(err));
//        return EBADF;
//    }
//
//    nvm_queue_t host_sq;
//    nvm_queue_t host_cq;
//    int status = create_queues(ioctl_fd, ctrl, dev, &host_cq, &host_sq);
//    if (status != 0)
//    {
//        fprintf(stderr, "Failed to create queues: %s\n", strerror(status));
//        return status;
//    }
//
//    nvm_queue_t dev_sq;
//    //nvm_queue_t dev_cq;
//
//    err = hipMalloc(&dev_sq, sizeof(struct nvm_queue));
//    if (err != hipSuccess)
//    {
//        fprintf(stderr, "Failed to allocate device memory: %s\n", hipGetErrorString(err));
//        return ENOMEM;
//    }
//
////    err = hipMalloc(&dev_cq, sizeof(struct nvm_queue));
////    if (err != hipSuccess)
////    {
////        hipFree(dev_sq);
////        fprintf(stderr, "Failed to allocate device memory: %s\n", hipGetErrorString(err));
////        return ENOMEM;
////    }
//
//    memory_t* host_buffer = get_gpu_buffer(ioctl_fd, dev, sizeof(uint32_t));
//    if (host_buffer == NULL)
//    {
//        hipFree(dev_sq);
//        //hipFree(dev_cq);
//        fprintf(stderr, "Failed to allocate buffer\n");
//        return ENOMEM;
//    }
//
//    memory_t* dev_buffer;
//    err = hipMalloc(&dev_buffer, sizeof(memory_t) + sizeof(uint64_t) * host_buffer->n_addrs);
//    if (err != hipSuccess)
//    {
//        put_gpu_buffer(ioctl_fd, host_buffer);
//        hipFree(dev_sq);
//        //hipFree(dev_cq);
//        fprintf(stderr, "Failed to allocate device memory: %s\n", hipGetErrorString(err));
//        return ENOMEM;
//    }
//
//    hipHostRegister(reg_ptr, reg_len, hipHostRegisterIoMemory);
//    
//    void* db;
//    hipHostGetDevicePointer(&db, reg_ptr, 0);
//    host_sq->db = SQ_DBL(db, host_sq->no, ctrl->dstrd);
//
//    //prepare_read_cmd(host_sq, 1, 512, host_buffer, 0, 1);
//    uint32_t result = 0xcafebabe;
//    //prepare_write_cmd(host_sq, 1, 512, host_buffer, 0, 1);
//    //uint32_t result = 0xdeadbeef;
//
//    hipMemcpy(dev_sq, host_sq, sizeof(struct nvm_queue), hipMemcpyHostToDevice);
//    //hipMemcpy(dev_cq, host_cq, sizeof(struct nvm_queue), hipMemcpyHostToDevice);
//    hipMemcpy(dev_buffer, host_buffer, sizeof(memory_t) + sizeof(uint64_t) * host_buffer->n_addrs, hipMemcpyHostToDevice);
//
//    //hipMemset(host_buffer->virt_addr, 0xca, sizeof(uint32_t));
//    hipMemcpy(host_buffer->virt_addr, &result, sizeof(uint32_t), hipMemcpyHostToDevice);
//
//    // this works on gpu too
//    //sq_submit(host_sq);
//
//    uint32_t* v;
//    hipMalloc(&v, sizeof(uint32_t));
//
//    do_work<<<1, 1>>>(dev_buffer, dev_sq, v);
//
//    // hack
//    usleep(5000000);
//
//    fprintf(stderr, "Polling...\n");
//    //struct completion* cpl = cq_dequeue_block(host_cq, ctrl);
////    if (cpl != NULL)
////    {
////        fprintf(stderr, "cid=%u sct=%x sc=%x\n", *CPL_CID(cpl), SCT(cpl), SC(cpl));
////    }
//
//    struct completion cpl;
//    memset(&cpl, 0xff, sizeof(cpl));
//    fprintf(stderr, "cid=%u sct=%x sc=%x\n", *CPL_CID(&cpl), SCT(&cpl), SC(&cpl));
//    hipMemcpy(&cpl, host_cq->page.virt_addr, sizeof(struct completion), hipMemcpyDeviceToHost);
//    fprintf(stderr, "cid=%u sct=%x sc=%x\n", *CPL_CID(&cpl), SCT(&cpl), SC(&cpl));
//    
//        
//    result = 0xfefefefe;
//    //hipMemcpy(&result, value, sizeof(uint32_t), hipMemcpyDeviceToHost);
//    hipMemcpy(&result, host_buffer->virt_addr, sizeof(uint32_t), hipMemcpyDeviceToHost);
//
//    fprintf(stderr, "%x\n", result);
//
//    hipMemcpy(&result, v, sizeof(uint32_t), hipMemcpyDeviceToHost);
//    fprintf(stderr, "%x\n", result);
//
//    hipFree(dev_buffer);
//    put_gpu_buffer(ioctl_fd, host_buffer);
//    hipFree(dev_sq);
////    hipFree(dev_cq);
//    return 0;
//}
//

