#include <hip/hip_runtime.h>
#include "types.h"
#include "queue.h"
#include "command.h"
#include "util.h"
#include <cstddef>
#include <cstdint>
#include <ctime>
#include <errno.h>


#define PHASE(p)    _RB(*CPL_STATUS(p),  0,  0) // Offset to phase tag bit


extern "C" __host__ __device__
struct command* sq_enqueue(nvm_queue_t* sq)
{
    // Check the capacity
    if ((sq->tail - sq->head) % sq->max_entries == sq->max_entries - 1)
    {
        return NULL;
    }

    // Get slot at end of queue
    struct command* ptr = 
        (struct command*) (((unsigned char*) sq->virt_addr) + sq->entry_size * sq->tail);

    // Increase tail pointer and wrap around if necessary
    if (++sq->tail >= sq->max_entries)
    {
        sq->tail = 0;
    }

    // Set command identifier to equal tail pointer
    // The caller may override this by manually setting the CID field in DWORD0
    *CMD_CID(ptr) = sq->tail;

    return ptr;
}


extern "C" __host__ __device__
struct completion* cq_poll(const nvm_queue_t* cq)
{
    struct completion* ptr = 
        (struct completion*) (((unsigned char*) cq->virt_addr) + cq->entry_size * cq->head);

    // Check if new completion is ready by checking the phase tag
    if (!!PHASE(ptr) != cq->phase)
    {
        return NULL;
    }

    return ptr;
}


extern "C" __host__ __device__
struct completion* cq_dequeue(nvm_queue_t* cq)
{
    struct completion* ptr = cq_poll(cq);

    if (ptr != NULL)
    {
        // Increase head pointer and wrap around if necessary
        if (++cq->head >= cq->max_entries)
        {
            cq->head = 0;
            cq->phase = !cq->phase;
        }
    }

    return ptr;
}


/* Delay execution by one millisecond */
__host__
static inline void delay(uint64_t& remaining_nsecs)
{
    if (remaining_nsecs == 0)
    {
        return;
    }

    timespec ts;
    ts.tv_sec = 0;
    ts.tv_nsec = _MIN(1000000UL, remaining_nsecs);

    clock_nanosleep(CLOCK_REALTIME, 0, &ts, NULL);

    remaining_nsecs -= _MIN(1000000UL, remaining_nsecs);
}


extern "C" __host__ 
struct completion* cq_dequeue_block(nvm_queue_t* cq, uint64_t timeout)
{
    uint64_t nsecs = timeout * 1000000UL;
    struct completion* cpl = cq_dequeue(cq);

    while (cpl == NULL && nsecs > 0)
    {
        delay(nsecs);
        cpl = cq_dequeue(cq);
    }

    return cpl;
}


extern "C" __host__ __device__
void sq_submit(const nvm_queue_t* sq)
{
    *((volatile uint32_t*) sq->db) = sq->tail;
}


extern "C" __host__ __device__
void cq_update(const nvm_queue_t* cq)
{
    *((volatile uint32_t*) cq->db) = cq->head;
}


extern "C" __host__ __device__
int sq_update(nvm_queue_t* sq, const struct completion* cpl)
{
    if (cpl == NULL)
    {
        return EAGAIN;
    }

    if (sq->no == *CPL_SQID(cpl))
    {
        // Update head pointer of submission queue
        sq->head = *CPL_SQHD(cpl);
        return 0;
    }

    return EBADF;
}

