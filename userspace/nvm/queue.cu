#include <hip/hip_runtime.h>
#include "types.h"
#include "queue.h"
#include "command.h"
#include "util.h"
#include <cstddef>
#include <cstdint>
#include <ctime>
#include <errno.h>


// Offset to phase tag bit
#define PHASE(p)            _RB(*CPL_STATUS(p),  0,  0)


extern "C" __host__ __device__
int cmd_data_ptr(struct command* cmd, memory_t* prp_list, memory_t* prps, size_t n_prps)
{
    cmd->dword[0] &= ~( (0x03 << 14) | (0x03 << 8) );

    if (n_prps > prps->n_addrs)
    {
        return ENOSPC;
    }

    cmd->dword[6] = (uint32_t) prps->bus_addr[0];
    cmd->dword[7] = (uint32_t) (prps->bus_addr[0] >> 32);

    if (n_prps <= 1)
    {
        cmd->dword[8] = 0;
        cmd->dword[9] = 0;
    }
    else if (n_prps == 2)
    {
        cmd->dword[8] = (uint32_t) prps->bus_addr[1];
        cmd->dword[9] = (uint32_t) (prps->bus_addr[1] >> 32);
    }
    else
    {
        // TODO Implement PRP list handling
    }

    return 0;
}


extern "C" __host__ __device__
void cmd_header(struct command* cmd, uint8_t opcode, uint32_t ns_id)
{
    cmd->dword[0] &= 0xffff0000;
    cmd->dword[0] |= (0x00 << 14) | (0x00 << 8) | (opcode & 0x7f);
    cmd->dword[1] = ns_id;
}


extern "C" __host__ __device__
struct command* sq_enqueue(nvm_queue_t sq)
{
    // Check the capacity
    if (sq->tail - sq->head >= sq->max_entries)
    {
        return NULL;
    }

    // Get slot at end of queue
    struct command* ptr = (struct command*) (((unsigned char*) sq->page.virt_addr) + sq->entry_size * sq->tail);

    // Increase tail pointer and wrap around if necessary
    if (++sq->tail >= sq->max_entries)
    {
        sq->tail = 0;
    }

    // Set command identifier to equal tail pointer
    // The caller may override this by manually setting the CID field in DWORD0
    *CMD_CID(ptr) = sq->tail;

    return ptr;
}


extern "C" __host__ __device__
struct completion* cq_poll(nvm_queue_t cq)
{
    struct completion* ptr = 
        (struct completion*) (((unsigned char*) cq->page.virt_addr) + cq->entry_size * cq->head);

    // Check if new completion is ready by checking the phase tag
    if (PHASE(ptr) != cq->phase)
    {
        return NULL;
    }

    return ptr;
}


extern "C" __host__ __device__
struct completion* cq_dequeue(nvm_queue_t cq, nvm_controller_t ctrl)
{
    struct completion* ptr = cq_poll(cq);

    if (ptr != NULL)
    {
        // Increase head pointer and wrap around if necessary
        if (++cq->head >= cq->max_entries)
        {
            cq->head = 0;
            cq->phase = !cq->phase;
        }

        if (ctrl != NULL)
        {
            // Update head pointer of submission queue
            ctrl->queues[*CPL_SQID(ptr)]->head = *CPL_SQHD(ptr);
        }
    }

    return ptr;
}


/* Delay execution by one millisecond */
__host__
static inline void delay(uint64_t& remaining_nsecs)
{
    if (remaining_nsecs == 0)
    {
        return;
    }

    timespec ts;
    ts.tv_sec = 0;
    ts.tv_nsec = _MIN(1000000UL, remaining_nsecs);

    clock_nanosleep(CLOCK_REALTIME, 0, &ts, NULL);

    remaining_nsecs -= _MIN(1000000UL, remaining_nsecs);
}


extern "C" __host__ 
struct completion* cq_dequeue_block(nvm_queue_t cq, nvm_controller_t ctrl)
{
    uint64_t nsecs = ctrl->timeout * 1000000UL;
    struct completion* cpl = cq_dequeue(cq, ctrl);

    while (cpl == NULL && nsecs > 0)
    {
        delay(nsecs);
        cpl = cq_dequeue(cq, ctrl);
    }

    return cpl;
}


extern "C" __host__ __device__
void sq_submit(nvm_queue_t sq)
{
    *((volatile uint32_t*) sq->db) = sq->tail;
}


extern "C" __host__ __device__
void cq_update(nvm_queue_t cq)
{
    *((volatile uint32_t*) cq->db) = cq->head;
}

