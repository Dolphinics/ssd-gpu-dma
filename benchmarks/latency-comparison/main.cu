#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvm_ctrl.h>
#include <nvm_types.h>
#include <nvm_queue.h>
#include <nvm_util.h>
#include <nvm_admin.h>
#include <nvm_error.h>
#include <nvm_cmd.h>
#include <string>
#include <stdexcept>
#include <cstdio>
#include <cstdint>
#include "ctrl.h"
#include "buffer.h"
#include "settings.h"
#ifdef __DIS_CLUSTER__
#include <sisci_api.h>
#endif

using error = std::runtime_error;
using std::string;


struct __align__(64) QueuePair
{
    size_t              pageSize;
    size_t              blockSize;
    uint32_t            nvmNamespace;
    size_t              maxDataSize;
    //void*               prpListPtr;
    //uint64_t            prpListAddr;
    nvm_queue_t         sq;
    nvm_queue_t         cq;
};


__device__ void movePage(volatile void* src, void* dst, size_t offset, uint16_t threadNum, size_t pageSize)
{
    uint8_t* source = ((uint8_t*) src) + (pageSize * threadNum);
    uint8_t* destination = ((uint8_t*) dst) + (pageSize * offset);

    for (size_t i = 0; i < pageSize; ++i)
    {
        destination[i] = source[i];
        source[i] = 0;
    }
}



__global__ void readPages(QueuePair* qp, uint64_t ioaddr, volatile void* src, void* dst, size_t numPages, uint64_t* errCount)
{
    const uint16_t numThreads = blockDim.x;
    const uint16_t threadNum = threadIdx.x;

    const size_t blocksPerPage = NVM_PAGE_TO_BLOCK(qp->pageSize, qp->blockSize, 1);
    uint64_t blockOffset = 0;  // FIXME

    if (threadNum == 0)
    {
        *errCount = 0;
    }
    __syncthreads();

    nvm_cmd_t* cmd = nullptr;
    for (size_t currPage = threadNum; currPage < numPages; currPage += numThreads)
    {
        size_t currBlock = NVM_PAGE_TO_BLOCK(qp->pageSize, qp->blockSize, currPage) + blockOffset;
        cmd = nvm_sq_enqueue_n(&qp->sq, cmd, numThreads, threadNum);

        nvm_cmd_header(cmd, NVM_IO_READ, qp->nvmNamespace);
        nvm_cmd_data_ptr(cmd, ioaddr + qp->pageSize * threadNum, 0);
        nvm_cmd_rw_blks(cmd, currBlock, blocksPerPage);

        if (threadNum == 0)
        {
            nvm_sq_submit(&qp->sq);

            for (uint16_t i = 0; i < numThreads; ++i)
            {
                nvm_cpl_t* cpl = nullptr;
                while ((cpl = nvm_cq_dequeue(&qp->cq)) == nullptr);
                nvm_sq_update(&qp->sq);

                if (!NVM_ERR_OK(cpl))
                {
                    *errCount = *errCount + 1;
                }
            }

            nvm_cq_update(&qp->cq);
        }
        //__threadfence();
        __syncthreads();

        movePage(src, dst, currPage, threadNum, qp->pageSize);
    }
}



static void prepareQueuePair(DmaPtr& qmem, QueuePair& qp, const Controller& ctrl, int device, uint32_t numThreads, uint32_t adapter, uint32_t id)
{
    size_t queueMemSize = ctrl.info.page_size * 2;
    //size_t prpListSize = ctrl.info.page_size * numThreads;
    size_t prpListSize = 0;

    // qmem->vaddr will be already a device pointer after the following call
    qmem = createDma(ctrl.ctrl, queueMemSize + prpListSize, device, adapter, id);

    qp.pageSize = ctrl.info.page_size;
    qp.blockSize = ctrl.ns.lba_data_size;
    qp.nvmNamespace = ctrl.ns.ns_id;
    qp.maxDataSize = ctrl.info.max_data_size;
    
    int status = nvm_admin_cq_create(ctrl.aq_ref, &qp.cq, 1, qmem->vaddr, qmem->ioaddrs[0]);
    if (!nvm_ok(status))
    {
        throw error(string("Failed to create completion queue: ") + nvm_strerror(status));
    }

    void* devicePtr = nullptr;
    hipError_t err = hipHostGetDevicePointer(&devicePtr, (void*) qp.cq.db, 0);
    if (err != hipSuccess)
    {
        throw err;
    }
    qp.cq.db = (volatile uint32_t*) devicePtr;

    status = nvm_admin_sq_create(ctrl.aq_ref, &qp.sq, &qp.cq, 1, NVM_DMA_OFFSET(qmem, 1), qmem->ioaddrs[1]);
    if (!nvm_ok(status))
    {
        throw error(string("Failed to create submission queue: ") + nvm_strerror(status));
    }

    err = hipHostGetDevicePointer(&devicePtr, (void*) qp.sq.db, 0);
    if (err != hipSuccess)
    {
        throw err;
    }
    qp.sq.db = (volatile uint32_t*) devicePtr;

   // qp.prpListPtr = NVM_DMA_OFFSET(qmem, 2);
   // qp.prpListAddr = qmem->ioaddrs[2];
}



static void verify(BufferPtr data, size_t size, const char* filename)
{
    auto buffer = createBuffer(size);

    hipError_t err = hipMemcpy(buffer.get(), data.get(), size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        throw error(string("Failed to copy data from destination: ") + hipGetErrorString(err));
    }

    // TODO: open filename for read and compare byte by byte

    FILE* fp = fopen(filename, "wb");
    fwrite(buffer.get(), 1, size, fp);
    fclose(fp);
}




static void use_nvm(const Controller& ctrl, const Settings& settings)
{
    uint32_t sid = settings.segmentId;

    DmaPtr queueMemory;
    QueuePair queuePair;
    prepareQueuePair(queueMemory, queuePair, ctrl, settings.cudaDevice, settings.numThreads, settings.adapter, sid++);

    auto deviceQueue = createBuffer(sizeof(QueuePair), settings.cudaDevice);
    auto err = hipMemcpy(deviceQueue.get(), &queuePair, sizeof(QueuePair), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        throw err;
    }

    const size_t pageSize = ctrl.info.page_size;
    const size_t blockSize = ctrl.ns.lba_data_size;

    size_t totalPages = NVM_PAGE_ALIGN(settings.numPages * pageSize, pageSize * settings.numThreads) / pageSize;
    size_t totalBlocks = NVM_PAGE_TO_BLOCK(pageSize, blockSize, totalPages);

    fprintf(stderr, "numThreads=%u, totalPages=%zu, totalBlocks=%zu\n",
            settings.numThreads, totalPages, totalBlocks);

    auto destination = createBuffer(pageSize * totalPages, settings.cudaDevice); // this is a host ptr
    
    auto source = createDma(ctrl.ctrl, pageSize * settings.numThreads, settings.cudaDevice, settings.adapter, sid++); // vaddr is a dev ptr

    uint64_t* ec = nullptr;
    err = hipMalloc(&ec, sizeof(uint64_t));
    if (err != hipSuccess)
    {
        throw err;
    }

    readPages<<<1, settings.numThreads>>>((QueuePair*) deviceQueue.get(), source->ioaddrs[0], source->vaddr, destination.get(), totalPages, ec);

    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        hipFree(ec);
        throw err;
    }

    uint64_t errorCount = 0;
    hipMemcpy(&errorCount, ec, sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipFree(ec);

    fprintf(stderr, "ec: %lx\n", errorCount);

    if (settings.verify != nullptr)
    {
        verify(destination, totalPages * pageSize, settings.verify);
    }
}



int main(int argc, char** argv)
{
#ifdef __DIS_CLUSTER__
    sci_error_t err;
    SCIInitialize(0, &err);
    if (err != SCI_ERR_OK)
    {
        fprintf(stderr, "Failed to initialize SISCI: %s\n", SCIGetErrorString(err));
        return 1;
    }
#endif

    Settings settings;

    try
    {
        if (argc != 2)
        {
            Controller ctrl(settings.controllerId, settings.nvmNamespace, settings.adapter, settings.segmentId++);
            ctrl.reserveQueues(1);

            hipError_t err = hipHostRegister((void*) ctrl.ctrl->mm_ptr, NVM_CTRL_MEM_MINSIZE, hipHostRegisterIoMemory);
            if (err != hipSuccess)
            {
                throw err;
            }

            use_nvm(ctrl, settings);

            hipHostUnregister((void*) ctrl.ctrl->mm_ptr);
        }
        else
        {
            //use_fd(grid, block, argv[1]);
        }
    }
    catch (const hipError_t err)
    {
        fprintf(stderr, "Unexpected CUDA error: %s\n", hipGetErrorString(err));
    }
    catch (const error& e)
    {
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }


#ifdef __DIS_CLUSTER__
    SCITerminate();
#endif
    return 0;
}
