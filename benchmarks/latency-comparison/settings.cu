#include <hip/hip_runtime.h>
#include "settings.h"

// TODO: copy settings from latency benchmark


Settings::Settings()
{
    cudaDevice = 0;
    blockDevicePath = nullptr;
    controllerPath = nullptr;
    controllerId = 0x80000;
    adapter = 0;
    segmentId = 0;
    nvmNamespace = 1;
    numPages = 4*4096;
    startBlock = 0;
    stats = false;
    verify = "readback.txt";
    numThreads = 16;
}
