#include <hip/hip_runtime.h>
#include "settings.h"

// TODO: copy settings from latency benchmark


Settings::Settings()
{
    cudaDevice = 0;
    blockDevicePath = nullptr;
    controllerPath = nullptr;
    controllerId = 0x80000;
    adapter = 0;
    segmentId = 0;
    nvmNamespace = 1;
    bufferLevel = 2;
    numChunks = 32;
    numPages = 16;
    startBlock = 0;
    stats = false;
    verify = "readback.txt";
    numThreads = 32;
}
