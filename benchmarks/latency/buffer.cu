#include <hip/hip_runtime.h>
#include <nvm_types.h>
#include <nvm_dma.h>
#include <nvm_util.h>
#include <nvm_error.h>
#include <memory>
#include <stdexcept>
#include <string>
#include <new>
#include <cstddef>
#include <cstdint>
#include <cstdlib>
#include "buffer.h"
#include <cstdio>

using error = std::runtime_error;
using std::string;



BufferPtr createBuffer(const nvm_ctrl_t* ctrl, uint32_t adapter, uint32_t id, size_t size, int dev)
{
    nvm_dma_t* dma = nullptr;
    void* bufferPtr = nullptr;
    void* devicePtr = nullptr;

    if (dev < 0)
    {
        return createBuffer(ctrl, adapter, id, size);
    }

    hipError_t err = hipSetDevice(dev);
    if (err != hipSuccess)
    {
        throw error(string("Failed to set CUDA device: ") + hipGetErrorString(err));
    }

    err = hipMalloc(&bufferPtr, size);
    if (err != hipSuccess)
    {
        throw error(string("Failed to allocate device memory: ") + hipGetErrorString(err));
    }

    hipPointerAttribute_t attrs;
    err = hipPointerGetAttributes(&attrs, bufferPtr);
    if (err != hipSuccess)
    {
        hipFree(bufferPtr);
        throw error(string("Failed to get pointer attributes: ") + hipGetErrorString(err));
    }

    devicePtr = attrs.devicePointer;

#ifdef __DIS_CLUSTER__
    int status = nvm_dis_dma_map_device(&dma, ctrl, adapter, id, devicePtr, size);
#else
    int status = nvm_dma_map_device(&dma, ctrl, devicePtr, size);
#endif
    if (!nvm_ok(status))
    {
        hipFree(bufferPtr);
        throw error(string("Failed to create local segment: ") + nvm_strerror(status));
    }

    dma->vaddr = bufferPtr;

    return BufferPtr(dma, [bufferPtr](nvm_dma_t* m) {
        nvm_dma_unmap(m);
        if (bufferPtr != nullptr)
        {
            hipFree(bufferPtr);
        }
    });
}



#ifdef __DIS_CLUSTER__
BufferPtr createBuffer(const nvm_ctrl_t* ctrl, uint32_t adapter, uint32_t id, size_t size)
{
    nvm_dma_t* dma = nullptr;

    int status = nvm_dis_dma_create(&dma, ctrl, adapter, id, size);
    if (!nvm_ok(status))
    {
        throw error(string("Failed to create local segment: ") + nvm_strerror(status));
    }

    return BufferPtr(dma, [](nvm_dma_t* m) { nvm_dma_unmap(m); });
}
#else
BufferPtr createBuffer(const nvm_ctrl_t* ctrl, uint32_t, uint32_t, size_t size)
{
    nvm_dma_t* dma = nullptr;
    void* ptr = nullptr;

    hipError_t err = hipHostAlloc(&ptr, size, hipHostMallocDefault);
    if (err != hipSuccess)
    {
        throw error(string("Failed to allocate memory: ") + hipGetErrorString(err));
    }

    int status = nvm_dma_map_host(&dma, ctrl, ptr, size);
    if (!nvm_ok(status))
    {
        hipHostFree(ptr);
        throw error(string("Failed to map host memory: ") + nvm_strerror(status));
    }

    return BufferPtr(dma, [ptr](nvm_dma_t* m) {
        nvm_dma_unmap(m);
        if (ptr != nullptr)
        {
            hipHostFree(ptr);
        }
    });
}
#endif



#ifdef __DIS_CLUSTER__
BufferPtr createRemoteBuffer(const nvm_ctrl_t* ctrl, uint32_t adapter, uint32_t segno, size_t size)
{
    nvm_dma_t* dma = nullptr;

    int status = nvm_dis_dma_connect(&dma, ctrl, adapter, segno, size, true); // FIXME: should be private
    if (!nvm_ok(status))
    {
        throw error(string("Failed to connect to segment: ") + nvm_strerror(status));
    }

    return BufferPtr(dma, [](nvm_dma_t* m) { nvm_dma_unmap(m); });
}
#endif

