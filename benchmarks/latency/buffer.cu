#include <hip/hip_runtime.h>
#include <nvm_types.h>
#include <nvm_dma.h>
#include <nvm_util.h>
#include <nvm_error.h>
#include <memory>
#include <stdexcept>
#include <string>
#include <new>
#include <cstddef>
#include <cstdint>
#include <cstdlib>
#include "buffer.h"
#include <cstdio>

using error = std::runtime_error;
using std::string;



BufferPtr createBuffer(const nvm_ctrl_t* ctrl, uint32_t adapter, uint32_t id, size_t size, int dev)
{
    nvm_dma_t* dma = nullptr;
    void* bufferPtr = nullptr;

    if (dev < 0)
    {
        return createBuffer(ctrl, adapter, id, size);
    }

    hipError_t err = hipSetDevice(dev);
    if (err != hipSuccess)
    {
        throw error(string("Failed to set CUDA device: ") + hipGetErrorString(err));
    }

    err = hipMalloc(&bufferPtr, size);
    if (err != hipSuccess)
    {
        throw error(string("Failed to allocate device memory: ") + hipGetErrorString(err));
    }

    hipPointerAttribute_t attrs;
    err = hipPointerGetAttributes(&attrs, bufferPtr);
    if (err != hipSuccess)
    {
        hipFree(bufferPtr);
        throw error(string("Failed to get pointer attributes: ") + hipGetErrorString(err));
    }

    fprintf(stderr, "bufferPtr=%p devicePointer=%p\n", bufferPtr, attrs.devicePointer);

#ifdef __DIS_CLUSTER__
    int status = nvm_dis_dma_map_device(&dma, ctrl, adapter, id, attrs.devicePointer, size);
#else
    int status = nvm_dma_map_device(&dma, ctrl, attrs.devicePointer, size);
#endif
    if (!nvm_ok(status))
    {
        hipFree(bufferPtr);
        throw error(string("Failed to create local segment: ") + nvm_strerror(status));
    }

    dma->vaddr = bufferPtr;

    return BufferPtr(dma, [bufferPtr](nvm_dma_t* m) {
        nvm_dma_unmap(m);
        if (bufferPtr != nullptr)
        {
            hipFree(bufferPtr);
        }
    });
}



#ifdef __DIS_CLUSTER__
BufferPtr createBuffer(const nvm_ctrl_t* ctrl, uint32_t adapter, uint32_t id, size_t size)
{
    nvm_dma_t* dma = nullptr;

    int status = nvm_dis_dma_create(&dma, ctrl, adapter, id, size);
    if (!nvm_ok(status))
    {
        throw error(string("Failed to create local segment: ") + nvm_strerror(status));
    }

    return BufferPtr(dma, [](nvm_dma_t* m) { nvm_dma_unmap(m); });
}
#else
BufferPtr createBuffer(const nvm_ctrl_t* ctrl, uint32_t, uint32_t, size_t size)
{
    nvm_dma_t* dma = nullptr;
    void* ptr = nullptr;

    int status = posix_memalign(&ptr, ctrl->page_size, size);
    if (status != 0)
    {
        throw error(string("Failed to allocate memory: ") + strerror(errno));
    }

    status = nvm_dma_map_host(&dma, ctrl, ptr, size);
    if (!nvm_ok(status))
    {
        free(ptr);
        throw error(string("Failed to map host memory: ") + nvm_strerror(status));
    }

    return BufferPtr(dma, [ptr](nvm_dma_t* m) {
        nvm_dma_unmap(m);
        if (ptr != nullptr)
        {
            free(ptr);
        }
    });
}
#endif



#ifdef __DIS_CLUSTER__
BufferPtr createRemoteBuffer(const nvm_ctrl_t* ctrl, uint32_t adapter, uint32_t segno, size_t size)
{
    nvm_dma_t* dma = nullptr;

    int status = nvm_dis_dma_connect(&dma, ctrl, adapter, segno, size, true); // FIXME: should be private
    if (!nvm_ok(status))
    {
        throw error(string("Failed to connect to segment: ") + nvm_strerror(status));
    }

    return BufferPtr(dma, [](nvm_dma_t* m) { nvm_dma_unmap(m); });
}
#endif

