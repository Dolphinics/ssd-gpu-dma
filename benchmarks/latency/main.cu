#include "hip/hip_runtime.h"
#include "settings.h"
#include "benchmark.h"
#include "buffer.h"
#include "ctrl.h"
#include "queue.h"
#include "barrier.h"
#include <nvm_types.h>
#include <nvm_ctrl.h>
#include <nvm_error.h>
#include <nvm_util.h>
#include <nvm_queue.h>
#include <nvm_cmd.h>
#include <stdexcept>
#include <vector>
#include <memory>
#include <algorithm>
#include <functional>
#include <thread>
#include <chrono>
#include <string>
#include <limits>
#include <cstring>
#include <cstdlib>
#include <cmath>
#ifdef __DIS_CLUSTER__
#include <sisci_api.h>
#endif

using std::string;
using std::runtime_error;
using std::make_shared;
using std::thread;



static string patternString(const Settings& settings)
{
    switch (settings.pattern)
    {
        case AccessPattern::LINEAR:
            return "linear";

        case AccessPattern::SEQUENTIAL:
            return "sequential";

        case AccessPattern::RANDOM_LINEAR:
            return "random-offset";

        case AccessPattern::RANDOM_CHUNK:
            return "random-chunks";

        case AccessPattern::RANDOM_PAGE:
            return "random-pages";
    }

    throw runtime_error("Unknown access pattern");
}



static size_t createQueues(const Controller& ctrl, Settings& settings, QueueList& queues)
{
    const size_t pageSize = ctrl.info.page_size;
    const size_t blockSize = ctrl.ns.lba_data_size;

    size_t pages = 0;
    size_t totalPages = NVM_PAGE_ALIGN(settings.numBlocks * blockSize, pageSize) / pageSize;

    srand(settings.startBlock);

    for (uint16_t no = 1; no <= ctrl.numQueues; ++no)
    {
#ifdef __DIS_CLUSTER__
        auto queue = make_shared<Queue>(ctrl, settings.adapter, settings.segmentId++, no, settings.queueDepth, settings.remote);
#else
        auto queue = make_shared<Queue>(ctrl, no, settings.queueDepth);
#endif

        switch (settings.pattern)
        {
            case AccessPattern::LINEAR:
                pages += prepareRange(queue->transfers, ctrl, settings.write, pages, settings.startBlock, settings.numBlocks);
                break;

            case AccessPattern::SEQUENTIAL:
                if (no == ctrl.numQueues)
                {
                    auto start = settings.startBlock + NVM_PAGE_TO_BLOCK(pageSize, blockSize, pages);
                    auto blocks = settings.numBlocks - NVM_PAGE_TO_BLOCK(pageSize, blockSize, pages);
                    pages += prepareRange(queue->transfers, ctrl, settings.write, pages, start, blocks);
                }
                else
                {
                    auto start = settings.startBlock + NVM_PAGE_TO_BLOCK(pageSize, blockSize, pages);
                    auto blocks = NVM_PAGE_TO_BLOCK(pageSize, blockSize, totalPages / ctrl.numQueues);
                    pages += prepareRange(queue->transfers, ctrl, settings.write, pages, start, blocks);
                }
                break;

            case AccessPattern::RANDOM_LINEAR:
                pages += fillRandom(queue->transfers, ctrl, settings.write, settings.numBlocks, false, false);
                break;

            case AccessPattern::RANDOM_CHUNK:
                pages += fillRandom(queue->transfers, ctrl, settings.write, settings.numBlocks, true, false);
                break;

            case AccessPattern::RANDOM_PAGE:
                pages += fillRandom(queue->transfers, ctrl, settings.write, settings.numBlocks, true, true);
                break;
        }

        if (queue->transfers.empty())
        {
            throw runtime_error("Transfer size is too small for more than one queue");
        }

        fprintf(stderr, "Queue #%02u %s %zu commands\n",
                no, settings.remote ? "remote" : "local", queue->transfers.size());

        queues.push_back(queue);
    }
    
    return pages;
}



static void copyMemory(const MemPtr& outputBuffer, const DmaPtr& buffer, size_t offset, size_t size, int cudaDevice)
{
    void* sourcePtr = (void*) (((unsigned char*) buffer->vaddr) + offset);

    if (cudaDevice != -1)
    {
        hipError_t err = hipMemcpy(outputBuffer.get(), sourcePtr, size, hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            throw runtime_error(string("Failed to copy memory from device: ") + hipGetErrorString(err));
        }
    }
    else
    {
        memcpy(outputBuffer.get(), sourcePtr, size);
    }
}



static void writeToFile(const MemPtr& outputBuffer, size_t size, const string& filename)
{
    FILE* fp = fopen(filename.c_str(), "wb");
    if (fp == nullptr)
    {
        throw runtime_error(string("Failed to open file: ") + strerror(errno));
    }

    fwrite(outputBuffer.get(), 1, size, fp);
    fflush(fp);
    fclose(fp);
}



static void outputFile(const Controller& ctrl, const QueueList& queues, const DmaPtr& buffer, const Settings& settings)
{
    size_t fileSize = settings.numBlocks * ctrl.ns.lba_data_size;

    if (settings.write)
    {
        throw runtime_error("Unable to verify written data");
    }

    auto outputBuffer = createHostMemory(fileSize);

    if (settings.pattern == AccessPattern::SEQUENTIAL)
    {
        copyMemory(outputBuffer, buffer, 0, fileSize, settings.cudaDevice);
        writeToFile(outputBuffer, fileSize, string(settings.filename) + "-sequential");
    }
    else if (settings.pattern == AccessPattern::LINEAR)
    {
        for (const auto& queue: queues)
        {
            size_t offset = NVM_PAGE_ALIGN(fileSize, ctrl.info.page_size) * (queue->no - 1);
            copyMemory(outputBuffer, buffer, offset, fileSize, settings.cudaDevice);
            writeToFile(outputBuffer, fileSize, string(settings.filename) + "-linear-q" + std::to_string(queue->no));
        }
    }
}



static Time sendWindow(QueuePtr& queue, TransferPtr& from, const TransferPtr& to, const DmaPtr& buffer, uint32_t ns, Barrier* barrier)
{
    size_t numCommands = 0;
    size_t numBlocks = 0;
    nvm_cmd_t local;
    memset(&local, 0, sizeof(local));

    // Fill up to queue depth with commands
    for (numCommands = 0; numCommands < queue->depth && from != to; ++numCommands, ++from)
    {
        nvm_cmd_t* cmd = nvm_sq_enqueue(&queue->sq);
        if (cmd == nullptr)
        {
            throw runtime_error(string("Queue is full, should not happen!"));
        }

        const Transfer& t = *from;
        void* prpListPtr = NVM_DMA_OFFSET(queue->sq_mem, 1 + numCommands);
        uint64_t prpListAddr = queue->sq_mem->ioaddrs[1 + numCommands];
        
        nvm_cmd_header(&local, NVM_DEFAULT_CID(&queue->sq), t.write ? NVM_IO_WRITE : NVM_IO_READ, ns);
        nvm_cmd_rw_blks(&local, t.startBlock, t.numBlocks);
        nvm_cmd_data(&local, buffer->page_size, t.numPages, prpListPtr, prpListAddr, &buffer->ioaddrs[t.startPage]);

        // Write command to remote memory in one go (due to write combining)
        *cmd = local;

        numBlocks += t.numBlocks;
    }

    // Sync with other threads
    barrier->wait();

    // Get current time before submitting
    auto before = std::chrono::high_resolution_clock::now();
    nvm_sq_submit(&queue->sq);
    std::this_thread::yield();

    // Wait for all completions
    for (size_t i = 0; i < numCommands; ++i)
    {
        nvm_cpl_t* cpl;
        while ((cpl = nvm_cq_dequeue(&queue->cq)) == nullptr)
        {
            std::this_thread::yield();
        }

        nvm_sq_update(&queue->sq);

        if (!NVM_ERR_OK(cpl))
        {
            fprintf(stderr, "%u: %s\n", queue->no, nvm_strerror(NVM_ERR_STATUS(cpl)));
        }

        nvm_cq_update(&queue->cq);
    }

    // Get current time after all commands completed
    auto after = std::chrono::high_resolution_clock::now();

    return Time(numCommands, numBlocks, after - before);
}



static void flush(QueuePtr& queue, uint32_t ns)
{
    nvm_cmd_t* cmd = nvm_sq_enqueue(&queue->sq);
    if (cmd == nullptr)
    {
        throw runtime_error(string("Queue is full, should not happen!"));
    }

    nvm_cmd_header(cmd, NVM_DEFAULT_CID(&queue->sq), NVM_IO_FLUSH, ns);
    nvm_cmd_data_ptr(cmd, 0, 0);

    nvm_sq_submit(&queue->sq);

    while (nvm_cq_dequeue(&queue->cq) == nullptr)
    {
        std::this_thread::yield();
    }
    nvm_sq_update(&queue->sq);
    nvm_cq_update(&queue->cq);
}



static void measureLatency(QueuePtr queue, const DmaPtr buffer, Times* times, const Settings& settings, Barrier* barrier)
{
    for (size_t i = 0; i < settings.repetitions; ++i)
    {
        const TransferPtr transferEnd = queue->transfers.cend();
        TransferPtr transferPtr = queue->transfers.cbegin();
        
        while (transferPtr != transferEnd)
        {
            auto time = sendWindow(queue, transferPtr, transferEnd, buffer, settings.nvmNamespace, barrier);

            times->push_back(time);
        }

        flush(queue, settings.nvmNamespace);

    }
}



static size_t consumeCompletions(QueuePtr& queue)
{
    nvm_queue_t* cq = &queue->cq;
    nvm_queue_t* sq = &queue->sq;

    nvm_cpl_t* cpl = nullptr;
    size_t numCpls = 0;

    while ((cpl = nvm_cq_dequeue(cq)) != nullptr)
    {
        nvm_sq_update(sq);

        if (!NVM_ERR_OK(cpl))
        {
            fprintf(stderr, "%u: %s\n", queue->no, nvm_strerror(NVM_ERR_STATUS(cpl)));
        }

        ++numCpls;
    }

    nvm_cq_update(cq);

    return numCpls;
}



static void measureBandwidth(QueuePtr queue, const DmaPtr buffer, Times* times, const Settings& settings, Barrier* barrier)
{
    nvm_queue_t* sq = &queue->sq;
    nvm_cmd_t local;
    memset(&local, 0, sizeof(local));

    barrier->wait();

    for (size_t i = 0; i < settings.repetitions; ++i)
    {
        size_t numCmds = 0;
        size_t numCpls = 0;
        size_t totalCmds = 0;
        size_t totalCpls = 0;
        size_t numBlocks = 0;

        auto before = std::chrono::high_resolution_clock::now();

        for (const auto& transfer: queue->transfers)
        {
            nvm_cmd_t* cmd = nullptr;

            while (numCmds == queue->depth || (cmd = nvm_sq_enqueue(sq)) == nullptr)
            {
                nvm_sq_submit(sq);
                std::this_thread::yield();
                numCpls = consumeCompletions(queue);
                numCmds -= numCpls;
                totalCpls += numCpls;
            }
            
            void* prpListPtr = NVM_DMA_OFFSET(queue->sq_mem, 1 + numCmds);
            uint64_t prpListAddr = queue->sq_mem->ioaddrs[1 + numCmds];

            nvm_cmd_header(&local, NVM_DEFAULT_CID(sq), transfer.write ? NVM_IO_WRITE : NVM_IO_READ, settings.nvmNamespace);
            nvm_cmd_rw_blks(&local, transfer.startBlock, transfer.numBlocks);
            nvm_cmd_data(&local, buffer->page_size, transfer.numPages, prpListPtr, prpListAddr, &buffer->ioaddrs[transfer.startPage]);
           
            // Command may be a pointer to remote memory,
            // so to avoid issues with write-combining we copy
            // everything in one go
            *cmd = local;

            numBlocks += transfer.numBlocks;

            ++numCmds;
            ++totalCmds;
        }

        nvm_sq_submit(sq);

        while (totalCpls != totalCmds)
        {
            std::this_thread::yield();
            numCpls = consumeCompletions(queue);
            totalCpls += numCpls;
            numCmds -= numCpls;
        }

        auto after = std::chrono::high_resolution_clock::now();
        times->push_back(Time(totalCmds, numBlocks, after - before));

        flush(queue, settings.nvmNamespace);
    }
}



static double percentile(const std::vector<double>& values, double p)
{
    double index = ceil(p * values.size());
    return values[index];
}


static void printPercentiles(std::vector<double>& data)
{
    std::sort(data.begin(), data.end(), std::greater<double>());
    std::reverse(data.begin(), data.end());

    fprintf(stderr, "count: %14zu\n", data.size());
    fprintf(stderr, "  max: %14.3f\n", data.back());
    for (auto p: {.99, .97, .95, .90, .75, .50, .25})
    {
        fprintf(stderr, " %4.2f: %14.3f\n", p, percentile(data, p));
    }
    fprintf(stderr, "  min: %14.3f\n", data.front());
}

static void printStats(const QueuePtr& queue, const Times& times, size_t blockSize, const Settings& settings)
{
    if (queue->no == 1)
    {
        fprintf(stdout, "#%5s; %6s; %4s; %13s; %8s; %8s; %12s; %12s; %12s;\n",
                "queue", "sqhost", "type", "pattern", "depth", "cmds", "blocks", "lat", "bw");
        fflush(stdout);
    }

    auto pattern = patternString(settings);
    auto patternPtr = pattern.c_str();

#ifdef __DIS_CLUSTER__
    bool local = !settings.remote;
#else
    bool local = true;
#endif

    bool gpu = settings.cudaDevice != -1;

    for (const auto& t: times)
    {
        auto time = t.time.count();
        auto blocks = t.blocks;
        
        double bw = (blocks * blockSize) / time;

        fprintf(stdout, " %5x; %6s; %4s; %13s; %8zu; %8u; %12zu; %12.3f; %12.3f;\n",
                queue->no, local ? "local" : "remote", gpu ? "gpu" : "ram", patternPtr,
                queue->depth, t.commands, t.blocks, time, bw);
        fflush(stdout);
    }
}

static void bandwidthStats(const Times& times, size_t blockSize, std::vector<double>& results)
{
    for (const auto& t: times)
    {
        const auto currentTime = t.time.count();
        const auto blocks = t.blocks;
        const double bw = (blocks * blockSize) / currentTime;
        results.push_back(bw);
    }
}


static void latencyStats(const Times& times, std::vector<double>& results)
{
    for (const auto& t: times)
    {
        results.push_back(t.time.count());
    }
}


static void benchmark(const QueueList& queues, const DmaPtr& buffer, const Settings& settings, size_t blockSize)
{
    hipError_t err;
    Times times[queues.size()];
    thread threads[queues.size()];

    if (settings.cudaDevice != -1)
    {
        err = hipSetDevice(settings.cudaDevice);
        if (err != hipSuccess)
        {
            throw runtime_error(string("Failed to set CUDA device: ") + hipGetErrorString(err));
        }

        auto size = settings.numBlocks * blockSize;
        err = hipMemset(buffer.get(), 0, size);
    }

    Barrier barrier(queues.size());

    if (settings.latency)
    {
        for (size_t i = 0; i < queues.size(); ++i)
        {
            Times* t = &times[i];
            QueuePtr q = queues[i];

            //auto func = std::bind(measureLatency, q, buffer, t, settings, &barrier);
            //threads[i] = thread(func);
            threads[i] = thread([q, buffer, t, settings, &barrier]() {
                measureLatency(q, buffer, t, settings, &barrier);
            });
        }
    }
    else
    {
        for (size_t i = 0; i < queues.size(); ++i)
        {
            Times* t = &times[i];
            QueuePtr q = queues[i];

            threads[i] = thread([q, buffer, t, settings, &barrier]() {
                measureBandwidth(q, buffer, t, settings, &barrier);
            });
        }
    }

    fprintf(stderr, "Running benchmark... (mode=%s, pattern=%s)\n", 
            settings.latency ? "latency" : "bandwidth", patternString(settings).c_str());

    std::vector<double> all;
    for (size_t i = 0; i < queues.size(); ++i)
    {
        threads[i].join();
        if (settings.latency)
        {
            latencyStats(times[i], all);
        }
        else
        {
            bandwidthStats(times[i], blockSize, all);
        }

        if (settings.stats)
        {
            printStats(queues[i], times[i], blockSize, settings);
        }
    }

    printPercentiles(all);

    if (settings.cudaDevice != -1)
    {
        err = hipDeviceSynchronize();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Synchronizing CUDA device failed: %s\n", hipGetErrorString(err));
        }
    }
}



int main(int argc, char** argv)
{
    Settings settings;

    // Parse command line arguments
    try
    {
        settings.parseArguments(argc, argv);
    }
    catch (const string& s)
    {
        fprintf(stderr, "%s\n", s.c_str());
        return 1;
    }

#ifdef __DIS_CLUSTER__
    sci_error_t err;
    SCIInitialize(0, &err);
    if (err != SCI_ERR_OK)
    {
        fprintf(stderr, "Something went wrong: %s\n", SCIGetErrorString(err));
        return 1;
    }

    sci_desc_t sd;
    SCIOpen(&sd, 0, &err);
    if (err != SCI_ERR_OK)
    {
        fprintf(stderr, "Something went wrong: %s\n", SCIGetErrorString(err));
        return 1;
    }

    sci_device_t cudaDev;
    if (settings.cudaDeviceId)
    {
        SCIBorrowDevice(sd, &cudaDev, settings.cudaDeviceId, SCI_FLAG_EXCLUSIVE, &err);
        if (err != SCI_ERR_OK)
        {
            fprintf(stderr, "Something went wrong: %s\n", SCIGetErrorString(err));
            return 1;
        }
    }
#endif

    try
    {
        fprintf(stderr, "Resetting controller...\n");
#ifdef __DIS_CLUSTER__
        Controller ctrl(settings.controllerId, settings.adapter, settings.segmentId++, settings.nvmNamespace, settings.numQueues);
#else
        Controller ctrl(settings.controllerPath, settings.nvmNamespace, settings.numQueues);
#endif

        settings.numQueues = ctrl.numQueues;

        QueueList queues;
        size_t numPages = createQueues(ctrl, settings, queues);

        fprintf(stderr, "Allocating %zu pages (%s)...\n", 
                numPages, settings.cudaDevice >= 0 ? "GPU" : "host");

        DmaPtr buffer;
        if (settings.cudaDevice != -1)
        {
            fprintf(stderr, "Using CUDA device: %d %s (%s)\n", settings.cudaDevice, settings.cudaDeviceName.c_str(), settings.getDeviceBDF().c_str());
            buffer = createDeviceDma(ctrl.ctrl, numPages * ctrl.ctrl->page_size, settings.cudaDevice, settings.adapter, settings.segmentId++);
        }
        else
        {
            buffer = createHostDma(ctrl.ctrl, numPages * ctrl.ctrl->page_size, settings.adapter, settings.segmentId++);
        }

        benchmark(queues, buffer, settings, ctrl.ns.lba_data_size);

        if (settings.filename != nullptr && !settings.write && !isRandom(settings.pattern))
        {
            fprintf(stderr, "Writing to file...\n");
            outputFile(ctrl, queues, buffer, settings);
        }
    }
    catch (const runtime_error& e)
    {
#ifdef __DIS_CLUSTER__
        if (settings.cudaDeviceId)
        {
            SCIReturnDevice(cudaDev, 0, &err);
        }
        SCIClose(sd, 0, &err);
#endif
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }

    fprintf(stderr, "OK!\n");

#ifdef __DIS_CLUSTER__
    if (settings.cudaDeviceId)
    {
        SCIReturnDevice(cudaDev, 0, &err);
    }
    SCIClose(sd, 0, &err);
    SCITerminate();
#endif
    return 0;
}

