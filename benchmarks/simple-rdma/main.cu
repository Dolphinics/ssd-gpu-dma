#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvm_types.h>
#include <nvm_ctrl.h>
#include <nvm_manager.h>
#include <nvm_rpc.h>
#include <memory>
#include <string>
#include <stdexcept>
#include <cstddef>
#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <getopt.h>
#include <errno.h>
#include "settings.h"
#include "dma.h"
#include "queue.h"
#include "transfer.h"
#include "benchmark.h"
#include "report.h"
#include "stats.h"


static void showUsage(const std::string& str)
{
    fprintf(stderr, "Usage: %s --ctrl=<ctrl id> --namespace=<ns id>\n",
            str.c_str());
}


static void showHelp(const std::string& str)
{
    showUsage(str);
}


static void parseOptions(int argc, char** argv, Settings& settings)
{
    static option options[] = {
        { "help", no_argument, nullptr, 'h' },
        { "ctrl", required_argument, nullptr, 'c' },
        { "device", required_argument, nullptr, 'd' },
        { "namespace", required_argument, nullptr, 'n' },
        { "queues", required_argument, nullptr, 'q' },
        { "blocks", required_argument, nullptr, 'b' },
        { "start", required_argument, nullptr, 's' },
        { "repeat", required_argument, nullptr, 'r' },
        { "chunk", required_argument, nullptr, 't' },
        { nullptr, false, nullptr, 0 }
    };

    std::string optstr = ":";
    for (auto opt : options)
    {
        optstr += opt.val;
        if (opt.has_arg)
        {
            optstr += ":";
        }
    }

    // Set default settings
    settings.cudaDevice = 0;
    settings.controllerId = 0;
    settings.nvmNamespace = 1;
    settings.numQueues = 1;
    settings.numBlocks = 0x1000;
    settings.startBlock = 0;
    settings.repeatLoops = 1000;
    settings.chunkSize = 0; // Use the controller's MDTS
    settings.blockSize = 0; // Figure this out later

    // Figure out how many CUDA devices available
    int numDevs = 0;
    hipError_t err = hipGetDeviceCount(&numDevs);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to get CUDA device count: %s\n", hipGetErrorString(err));
        exit(1);
    }

    // Parse options
    int optionsIdx = 0;
    char* endptr = nullptr;
    int option;
    while ((option = getopt_long(argc, argv, optstr.c_str(), options, &optionsIdx)) != -1)
    {
        switch (option)
        {
            case '?': // Unknown option
                fprintf(stderr, "Unknown option: `%s'\n", argv[optind - 1]);
                showUsage(argv[0]);
                exit(1);

            case ':': // Missing option argument
                fprintf(stderr, "Missing option argument: `%s'\n", argv[optind - 1]);
                showUsage(argv[0]);
                exit(1);

            case 'h': // Show help
                showHelp(argv[0]);
                exit(1);

            case 'c': // Set controller ID
                endptr = nullptr;
                settings.controllerId = strtoul(optarg, &endptr, 10);
                if (endptr == nullptr || *endptr != '\0')
                {
                    fprintf(stderr, "Invalid NVM controller ID: `%s'\n", optarg);
                    exit(1);
                }
                break;

            case 'd': // Set CUDA device
                endptr = nullptr;
                settings.cudaDevice = strtol(optarg, &endptr, 10);
                if (endptr == nullptr || *endptr != '\0' || settings.cudaDevice < 0 || settings.cudaDevice >= numDevs)
                {
                    fprintf(stderr, "Invalid CUDA device: `%s'\n", optarg);
                    exit(1);
                }
                break;

            case 'n': // Set NVM namespace
                endptr = nullptr;
                settings.nvmNamespace = strtoul(optarg, &endptr, 10);
                if (endptr == nullptr || *endptr != '\0' || settings.nvmNamespace == 0)
                {
                    fprintf(stderr, "Invalid NVM namespace: `%s'\n", optarg);
                    exit(1);
                }
                break;

            case 'q': // Set number of submission queues
                endptr = nullptr;
                settings.numQueues = strtoul(optarg, &endptr, 0);
                if (endptr == nullptr || *endptr != '\0' || settings.numQueues == 0)
                {
                    fprintf(stderr, "Invalid number of SQs: `%s'\n", optarg);
                    exit(1);
                }
                break;

            case 'b': // Set number of blocks
                endptr = nullptr;
                settings.numBlocks = strtoul(optarg, &endptr, 0);
                if (endptr == nullptr || *endptr != '\0' || settings.numBlocks == 0)
                {
                    fprintf(stderr, "Invalid number of disk blocks: `%s'\n", optarg);
                    exit(1);
                }
                break;

            case 's': // Set start block
                endptr = nullptr;
                settings.startBlock = strtoul(optarg, &endptr, 0);
                if (endptr == nullptr || *endptr != '\0')
                {
                    fprintf(stderr, "Invalid disk block: `%s'\n", optarg);
                    exit(1);
                }
                break;

            case 'r': // Set number of repeat loops for calculating average
                endptr = nullptr;
                settings.repeatLoops = strtoul(optarg, &endptr, 0);
                if (endptr == nullptr || *endptr != '\0' || settings.repeatLoops == 0)
                {
                    fprintf(stderr, "Invalid number of repeat loops: `%s'\n", optarg);
                    exit(1);
                }
                break;

            case 't': // Set chunk size
                endptr = nullptr;
                settings.chunkSize = strtoul(optarg, &endptr, 0);
                if (endptr == nullptr || *endptr != '\0')
                {
                    fprintf(stderr, "Invalid maximum transfer size (chunk size): `%s'\n", optarg);
                    exit(1);
                }
                break;

            default:
                if (optionsIdx != 0)
                {
                    fprintf(stderr, "Option `--%s' not implemented!\n", options[optionsIdx].name);
                }
                else
                {
                    fprintf(stderr, "Option `-%c' not implemented!\n", option);
                }
                exit(2);
        }
    }
}


static void identify(nvm_rpc_t rpc, nvm_ctrl_t ctrl, Settings& settings)
{
    auto page_buffer = createHostBuffer(ctrl, std::max((size_t) 0x1000, ctrl->page_size));

    nvm_ctrl_info_t ci;
    int err = nvm_rpc_ctrl_info(&ci, rpc, ctrl, (*page_buffer)->vaddr, (*page_buffer)->ioaddrs[0]);
    if (err != 0)
    {
        throw std::runtime_error("Failed to identify controller");
    }

    nvm_ns_info_t ni;
    err = nvm_rpc_ns_info(&ni, rpc, settings.nvmNamespace, (*page_buffer)->vaddr, (*page_buffer)->ioaddrs[0]);
    if (err != 0)
    {
        throw std::runtime_error("Failed to identify namespace");
    }

    settings.chunkSize = std::min(ci.max_transfer_size, settings.chunkSize);
    if (settings.chunkSize == 0)
    {
        settings.chunkSize = ci.max_transfer_size;
    }
    settings.blockSize = ni.lba_data_size;
}


static void direct(nvm_ctrl_t controller, QueueList& queues, const Settings& settings, std::vector<uint64_t>& times)
{
    report("Creating device buffer");
    auto buffer(createDeviceBuffer(controller, settings.numBlocks * settings.blockSize, settings.cudaDevice));
    report(true);

    report("Preparing transfer descriptors directly to GPU");
    TransferList transfers;
    prepareTransfers(transfers, controller, queues, buffer, settings);
    report(true);

#if (!defined(NDEBUG) && defined(DEBUG))
    report("Verifying transfer descriptors");
    controlTransferMemory(transfers, buffer);
    report(true);
#endif

    report("Reading from disk directly to GPU");
    for (size_t i = 0; i < settings.repeatLoops; ++i)
    {
        uint64_t time = benchmark(queues, transfers);
        times.push_back(time);
    }
    report(true);
}


static void bounce(nvm_ctrl_t controller, QueueList& queues, const Settings& settings, std::vector<uint64_t>& ramTimes, std::vector<uint64_t>& gpuTimes)
{
    report("Creating host buffer");
    auto buffer(createHostBuffer(controller, settings.numBlocks * settings.blockSize));
    report(true);

    report("Creating device buffer");
    hipError_t err = hipSetDevice(settings.cudaDevice);
    if (err != hipSuccess)
    {
        throw std::runtime_error("Failed to set CUDA device");
    }

    void* devicePointer = nullptr;
    err = hipMalloc(&devicePointer, settings.numBlocks * settings.blockSize);
    if (err != hipSuccess)
    {
        throw std::runtime_error("Failed to allocate device buffer: " + std::string(hipGetErrorString(err)));
    }
    report(true);

    report("Preparing transfer descriptors via RAM");
    TransferList transfers;
    prepareTransfers(transfers, controller, queues, buffer, settings);
    report(true);

#if (!defined(NDEBUG) && defined(DEBUG))
    report("Verifying transfer descriptors");
    controlTransferMemory(transfers, buffer);
    report(true);
#endif

    report("Reading from disk to GPU via RAM");
    for (size_t i = 0; i < settings.repeatLoops; ++i)
    {
        uint64_t time = benchmark(queues, transfers);

        uint64_t before = currentTime();
        err = hipMemcpy(devicePointer, (*buffer)->vaddr, settings.numBlocks * settings.blockSize, hipMemcpyHostToDevice);
        uint64_t after = currentTime();

        if (err != hipSuccess)
        {
            throw std::runtime_error("Failed to copy to device memory: " + std::string(hipGetErrorString(err)));
        }

        ramTimes.push_back(time);
        time += after - before;
        gpuTimes.push_back(time);
    }
    report(true);
}


int main(int argc, char** argv)
{
    Settings settings;
    parseOptions(argc, argv, settings);

    report("Setting CUDA device");
    hipError_t err = hipSetDevice(settings.cudaDevice);
    report(err);
    if (err != hipSuccess)
    {
        return 1;
    }

    // Create NVM controller reference
    nvm_ctrl_t controller;
    report("Getting controller reference");
    int nvmerr = nvm_ctrl_init(&controller, settings.controllerId);
    report(nvmerr);
    if (err != 0)
    {
        return 2;
    }

    // Allocate memory for ACQ and ASQ
    DmaPtr adminQueues;
    try
    {
        report("Creating admin queues");
        adminQueues = createHostBuffer(controller, 2 * controller->page_size);
        report(true);
    }
    catch (const std::runtime_error& err)
    {
        nvm_ctrl_free(controller);
        report(err);
        return 2;
    }

    // Reset NVM controller and configure admin queues
    nvm_manager_t manager = nullptr;
    report("Resetting controller");
    nvmerr = nvm_manager_register(&manager, controller, *adminQueues.get());
    report(nvmerr);
    if (nvmerr != 0)
    {
        nvm_ctrl_free(controller);
        return 2;
    }

    // Create RPC reference
    nvm_rpc_t rpcRef = nullptr;
    nvm_rpc_bind_local(&rpcRef, manager);

    // Identify controller and create IO queues
    QueueList queues;
    DmaPtr queueMemory;
    try
    {
        report("Identifying controller and namespace");
        identify(rpcRef, controller, settings);
        report(true);

        report("Creating IO queues");
        queueMemory = createHostBuffer(controller, (settings.numQueues + 1) * controller->page_size);
        
        createQueues(rpcRef, controller, queueMemory, queues);
        report(true);
    }
    catch (const std::runtime_error& err)
    {
        nvm_rpc_unbind(rpcRef);
        nvm_manager_unregister(manager);
        nvm_ctrl_free(controller);

        report(err);
        return 2;
    }

    // Run benchmark
    try
    {
        std::vector<uint64_t> gpuTimes;
        std::vector<uint64_t> ramTimes;

        bounce(controller, queues, settings, ramTimes, gpuTimes);

        showStatistics(settings, "RAM", ramTimes);
        printf("\n");

        showStatistics(settings, "Via RAM", gpuTimes);
        printf("\n");

        gpuTimes.clear();
        direct(controller, queues, settings, gpuTimes);
        showStatistics(settings, "GPU direct", gpuTimes);
    }
    catch (const std::runtime_error& err)
    {
        nvm_rpc_unbind(rpcRef);
        nvm_manager_unregister(manager);
        nvm_ctrl_free(controller);
        
        report(err);
        return 3;
    }
    
    // Release stuff and quit
    nvm_rpc_unbind(rpcRef);
    nvm_manager_unregister(manager);
    nvm_ctrl_free(controller);
    hipDeviceReset(); // for cuda-memcheck
    return 0;
}

