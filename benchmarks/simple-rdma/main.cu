#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvm_types.h>
#include <nvm_ctrl.h>
#include <nvm_manager.h>
#include <nvm_rpc.h>
#include <memory>
#include <string>
#include <stdexcept>
#include <algorithm>
#include <cstddef>
#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <getopt.h>
#include <errno.h>
#include "settings.h"
#include "dma.h"
#include "queue.h"
#include "transfer.h"
#include "benchmark.h"


static void showUsage(const std::string& str)
{
    fprintf(stderr, "Usage: %s --ctrl=<ctrl id> --namespace=<ns id>\n",
            str.c_str());
}


static void showHelp(const std::string& str)
{
    showUsage(str);
}


static void parseOptions(int argc, char** argv, Settings& settings)
{
    static option options[] = {
        { "help", no_argument, nullptr, 'h' },
        { "ctrl", required_argument, nullptr, 'c' },
        { "device", required_argument, nullptr, 'd' },
        { "namespace", required_argument, nullptr, 'n' },
        { "queues", required_argument, nullptr, 'q' },
        { "blocks", required_argument, nullptr, 'b' },
        { "start", required_argument, nullptr, 's' },
        { "repeat", required_argument, nullptr, 'r' },
        { "chunk", required_argument, nullptr, 't' },
        { nullptr, false, nullptr, 0 }
    };

    std::string optstr = ":";
    for (auto opt : options)
    {
        optstr += opt.val;
        if (opt.has_arg)
        {
            optstr += ":";
        }
    }

    // Set default settings
    settings.cudaDevice = 0;
    settings.controllerId = 0;
    settings.nvmNamespace = 1;
    settings.numQueues = 1;
    settings.numBlocks = 0x1000;
    settings.startBlock = 0;
    settings.repeatLoops = 1000;
    settings.chunkSize = 0; // Use the controller's MDTS
    settings.blockSize = 0; // Figure this out later

    // Figure out how many CUDA devices available
    int numDevs = 0;
    hipError_t err = hipGetDeviceCount(&numDevs);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to get CUDA device count: %s\n", hipGetErrorString(err));
        exit(1);
    }

    // Parse options
    int optionsIdx = 0;
    char* endptr = nullptr;
    int option;
    while ((option = getopt_long(argc, argv, optstr.c_str(), options, &optionsIdx)) != -1)
    {
        switch (option)
        {
            case '?': // Unknown option
                fprintf(stderr, "Unknown option: `%s'\n", argv[optind - 1]);
                showUsage(argv[0]);
                exit(1);

            case ':': // Missing option argument
                fprintf(stderr, "Missing option argument: `%s'\n", argv[optind - 1]);
                showUsage(argv[0]);
                exit(1);

            case 'h': // Show help
                showHelp(argv[0]);
                exit(1);

            case 'c': // Set controller ID
                endptr = nullptr;
                settings.controllerId = strtoul(optarg, &endptr, 10);
                if (endptr == nullptr || *endptr != '\0')
                {
                    fprintf(stderr, "Invalid NVM controller ID: `%s'\n", optarg);
                    exit(1);
                }
                break;

            case 'd': // Set CUDA device
                endptr = nullptr;
                settings.cudaDevice = strtol(optarg, &endptr, 10);
                if (endptr == nullptr || *endptr != '\0' || settings.cudaDevice < 0 || settings.cudaDevice >= numDevs)
                {
                    fprintf(stderr, "Invalid CUDA device: `%s'\n", optarg);
                    exit(1);
                }
                break;

            case 'n': // Set NVM namespace
                endptr = nullptr;
                settings.nvmNamespace = strtoul(optarg, &endptr, 10);
                if (endptr == nullptr || *endptr != '\0' || settings.nvmNamespace == 0)
                {
                    fprintf(stderr, "Invalid NVM namespace: `%s'\n", optarg);
                    exit(1);
                }
                break;

            case 'q': // Set number of submission queues
                endptr = nullptr;
                settings.numQueues = strtoul(optarg, &endptr, 0);
                if (endptr == nullptr || *endptr != '\0' || settings.numQueues == 0)
                {
                    fprintf(stderr, "Invalid number of SQs: `%s'\n", optarg);
                    exit(1);
                }
                break;

            case 'b': // Set number of blocks
                endptr = nullptr;
                settings.numBlocks = strtoul(optarg, &endptr, 0);
                if (endptr == nullptr || *endptr != '\0' || settings.numBlocks == 0)
                {
                    fprintf(stderr, "Invalid number of disk blocks: `%s'\n", optarg);
                    exit(1);
                }
                break;

            case 's': // Set start block
                endptr = nullptr;
                settings.startBlock = strtoul(optarg, &endptr, 0);
                if (endptr == nullptr || *endptr != '\0')
                {
                    fprintf(stderr, "Invalid disk block: `%s'\n", optarg);
                    exit(1);
                }
                break;

            case 'r': // Set number of repeat loops for calculating average
                endptr = nullptr;
                settings.repeatLoops = strtoul(optarg, &endptr, 0);
                if (endptr == nullptr || *endptr == '\0' || settings.repeatLoops == 0)
                {
                    fprintf(stderr, "Invalid number of repeat loops: `%s'\n", optarg);
                    exit(1);
                }
                break;

            case 't': // Set chunk size
                endptr = nullptr;
                settings.chunkSize = strtoul(optarg, &endptr, 0);
                if (endptr == nullptr || *endptr != '\0')
                {
                    fprintf(stderr, "Invalid maximum transfer size (chunk size): `%s'\n", optarg);
                    exit(1);
                }
                break;

            default:
                if (optionsIdx != 0)
                {
                    fprintf(stderr, "Option `--%s' not implemented!\n", options[optionsIdx].name);
                }
                else
                {
                    fprintf(stderr, "Option `-%c' not implemented!\n", option);
                }
                exit(2);
        }
    }
}


static void identify(nvm_rpc_t rpc, nvm_ctrl_t ctrl, Settings& settings)
{
    auto page_buffer = createHostBuffer(ctrl, std::max((size_t) 0x1000, ctrl->page_size));

    nvm_ctrl_info_t ci;
    int err = nvm_rpc_ctrl_info(&ci, rpc, ctrl, (*page_buffer)->vaddr, (*page_buffer)->ioaddrs[0]);
    if (err != 0)
    {
        throw std::runtime_error("Failed to identify controller");
    }

    nvm_ns_info_t ni;
    err = nvm_rpc_ns_info(&ni, rpc, settings.nvmNamespace, (*page_buffer)->vaddr, (*page_buffer)->ioaddrs[0]);
    if (err != 0)
    {
        throw std::runtime_error("Failed to identify namespace");
    }

    settings.chunkSize = std::min(ci.max_transfer_size, settings.chunkSize);
    if (settings.chunkSize == 0)
    {
        settings.chunkSize = ci.max_transfer_size;
    }
    settings.blockSize = ni.lba_data_size;
}


static void launch_benchmark(nvm_ctrl_t controller, QueueList& queues, DmaPtr hostBuffer, DmaPtr deviceBuffer, const Settings& settings)
{
        TransferList bouncedTransfers;
        prepareTransfers(bouncedTransfers, controller, queues, hostBuffer, settings);

        TransferList directTransfers;
        prepareTransfers(directTransfers, controller, queues, deviceBuffer, settings);

//        double bounceTimes[settings.repeatLoops];
//        double directTimes[settings.repeatLoops];
//
//        for (size_t i = 0; i < settings.repeatLoops; ++i)
//        {
//            bounceTimes[i] = benchmark(queues, bouncedTransfers, hostBuffer, deviceBuffer);
//            fprintf(stdout, "Bounce %.3f MiB/s\n", bounceTimes[i]);
//        }
//
//        for (size_t i = 0; i < settings.repeatLoops; ++i)
//        {
//            directTimes[i] = benchmark(queues, directTransfers, deviceBuffer);
//            fprintf(stdout, "Direct %.3f MiB/s\n", directTimes[i]);
//        }
}


int main(int argc, char** argv)
{
    Settings settings;
    parseOptions(argc, argv, settings);

    hipError_t err = hipSetDevice(settings.cudaDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to set CUDA device: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Create NVM controller reference
    nvm_ctrl_t controller;
    int nvmerr = nvm_ctrl_init(&controller, settings.controllerId);
    if (err != 0)
    {
        fprintf(stderr, "Failed to create controller reference: %s\n", strerror(nvmerr));
        return 2;
    }

    // Allocate memory for ACQ and ASQ
    DmaPtr adminQueues;
    try
    {
        adminQueues = createHostBuffer(controller, 2 * controller->page_size);
    }
    catch (const std::runtime_error& err)
    {
        nvm_ctrl_free(controller);
        fprintf(stderr, "%s\n", err.what());
        return 2;
    }

    // Reset NVM controller and configure admin queues
    nvm_manager_t manager = nullptr;
    nvmerr = nvm_manager_register(&manager, controller, *adminQueues.get());
    if (nvmerr != 0)
    {
        nvm_ctrl_free(controller);
        fprintf(stderr, "Failed to initialize admin queue manager: %s\n", strerror(nvmerr));
        return 2;
    }

    // Create RPC reference
    nvm_rpc_t rpcRef = nullptr;
    nvm_rpc_bind_local(&rpcRef, manager);

    // Identify controller and create IO queues
    QueueList queues;
    DmaPtr queueMemory;
    try
    {
        identify(rpcRef, controller, settings);

        queueMemory = createHostBuffer(controller, (settings.numQueues + 1) * controller->page_size);
        
        createQueues(rpcRef, controller, queueMemory, queues);
    }
    catch (const std::runtime_error& err)
    {
        nvm_rpc_unbind(rpcRef);
        nvm_manager_unregister(manager);
        nvm_ctrl_free(controller);

        fprintf(stderr, "%s\n", err.what());
        return 2;
    }

    // Run benchmark
    try
    {
        auto hostBuffer = createHostBuffer(controller, settings.numBlocks * settings.blockSize);

        auto deviceBuffer = createDeviceBuffer(controller, settings.numBlocks * settings.blockSize, settings.cudaDevice);

        launch_benchmark(controller, queues, hostBuffer, deviceBuffer, settings);

    }
    catch (const std::runtime_error& err)
    {
        nvm_rpc_unbind(rpcRef);
        nvm_manager_unregister(manager);
        nvm_ctrl_free(controller);

        fprintf(stderr, "%s\n", err.what());
        return 3;
    }
    
    // Release stuff and quit
    nvm_rpc_unbind(rpcRef);
    nvm_manager_unregister(manager);
    nvm_ctrl_free(controller);
    hipDeviceReset(); // for cuda-memcheck
    return 0;
}

