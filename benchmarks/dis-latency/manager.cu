#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdexcept>
#include <thread>
#include <mutex>
#include <vector>
#include <condition_variable>
#include <cstring>
#include <cerrno>
#include <ctime>
#include <cstdlib>
#include <csignal>
#include <sys/time.h>
#include <nvm_types.h>
#include <nvm_ctrl.h>
#include <nvm_manager.h>
#include <nvm_rpc.h>
#include <nvm_util.h>
#include <nvm_queue.h>
#include <nvm_command.h>
#include "settings.h"
#include "segment.h"
#include "transfer.h"
#include <sisci_api.h>

using std::cerr;
using std::cout;
using std::endl;
using std::runtime_error;



static uint16_t randomId()
{
    static uint16_t uniqueCounter = 0;

    timeval tv;
    srand(time(NULL));

    if (gettimeofday(&tv, NULL) == 0)
    {
        srand(tv.tv_usec);
    }

    return (rand() + uniqueCounter++) & 0xffff;
}


static uint64_t randomBlock(size_t max)
{
    return rand() & (max - 1);
}


static uint64_t currentTime()
{
    timespec ts;

    if (clock_gettime(CLOCK_REALTIME, &ts) < 0)
    {
        throw runtime_error(strerror(errno));
    }

    return ts.tv_sec * 1e6 + ts.tv_nsec / 1e3;
}


static void transfer(nvm_queue_t& cq, nvm_queue_t& sq, nvm_ctrl_t controller, const Settings& settings, bool warmup)
{
    auto dataSegment(createSegment(randomId(), 128 * controller->page_size));
    auto prpListSegment(createSegment(randomId(), controller->page_size));

    DmaPtr dataMap(createDmaMapping(dataSegment, controller, settings.ctrlAdapter));
    DmaPtr prpMap(createDmaMapping(prpListSegment, controller, settings.ctrlAdapter));

    for (size_t size: settings.transferSizes)
    {
        for (size_t i = 0; i < settings.repetitions; ++i)
        {
            nvm_cmd_t* cmd = sq_enqueue(&sq);
            nvm_cmd_header(cmd, NVM_IO_READ, settings.nvmNamespace);

            setDataPointer(cmd, dataMap, prpMap, settings.blockSize, size);

            uint16_t numBlocks = (size / settings.blockSize) + (size % settings.blockSize != 0);

            uint64_t block = randomBlock(settings.maxSize); 
            cmd->dword[10] = block;
            cmd->dword[11] = block >> 32;
            cmd->dword[12] = numBlocks - 1;

            uint64_t before = currentTime();
            sq_submit(&sq);

            nvm_cpl_t* cpl = nullptr;
            while ((cpl = cq_dequeue(&cq)) == nullptr);

            uint64_t after = currentTime();

            if (!CPL_OK(cpl))
            {
                throw runtime_error("Command failed");
            }

            sq_update_unchecked(&sq);
            cq_update(&cq);

            if (!warmup)
            {
                cout << size << "\t" << (after - before) << "\t" << block << "\t" << numBlocks << endl;
            }
        }
    }
}



static void runManager(nvm_ctrl_t controller, nvm_rpc_t rpc, const Settings& settings)
{
    nvm_queue_t cq;
    nvm_queue_t sq;

    SegmentPtr ioQueueMem(createSegment(randomId(), controller->page_size * 2));
    DmaPtr queueWindow(createDmaMapping(ioQueueMem, controller, settings.ctrlAdapter));
    memset((*queueWindow)->vaddr, 0, (*queueWindow)->n_ioaddrs * (*queueWindow)->page_size);

    int nvmerr = nvm_rpc_set_num_queues(rpc, 1, 1);
    if (nvmerr != 0)
    {
        throw runtime_error("Failed to set number of queues");
    }

    nvmerr = nvm_rpc_cq_create(&cq, rpc, controller, 1, (*queueWindow)->vaddr, (*queueWindow)->ioaddrs[0]);
    if (nvmerr != 0)
    {
        throw runtime_error("Failed to create CQ");
    }

    nvmerr = nvm_rpc_sq_create(&sq, rpc, controller, &cq, 1, 
            DMA_VADDR((*queueWindow)->vaddr, (*queueWindow)->page_size, 1), (*queueWindow)->ioaddrs[1]);
    if (nvmerr != 0)
    {
        throw runtime_error("Failed to create SQ");
    }

    transfer(cq, sq, controller, settings, true);
    transfer(cq, sq, controller, settings, false);
}


int main()
{
    Settings settings;

    nvm_ctrl_t controller;
    sci_error_t err;

    SCIInitialize(0, &err);

    int nvmerr = nvm_dis_ctrl_init(&controller, settings.ctrlId, settings.ctrlAdapter);
    if (nvmerr != 0)
    {
        cerr << "Failed to get controller reference: " << std::strerror(nvmerr) << endl;
        return 1;
    }

    SegmentPtr queueMemory;
    DmaPtr mapping;
    try
    {
        queueMemory = createSegment(randomId(), 2 * controller->page_size);
        mapping = createDmaMapping(queueMemory, controller, settings.ctrlAdapter);
        memset((*mapping)->vaddr, 0, (*mapping)->n_ioaddrs * (*mapping)->page_size);
    }
    catch (const runtime_error& err)
    {
        nvm_ctrl_free(controller);
        cerr << err.what() << endl;
        return 1;
    }

    nvm_manager_t manager = nullptr;
    nvmerr = nvm_manager_register(&manager, controller, *mapping.get());
    if (nvmerr != 0)
    {
        nvm_ctrl_free(controller);
        cerr << "Failed to reset controller" << endl;
        return 1;
    }

    nvm_rpc_t rpcRef = nullptr;
    nvm_rpc_bind_local(&rpcRef, manager);
    try
    {
        runManager(controller, rpcRef, settings);
    }
    catch (const runtime_error& err)
    {
        nvm_rpc_unbind(rpcRef);
        nvm_manager_unregister(manager);
        nvm_ctrl_free(controller);

        cerr << err.what() << endl;
        return 1;
    }

    nvm_rpc_unbind(rpcRef);
    nvm_manager_unregister(manager);
    nvm_ctrl_free(controller);
    SCITerminate();
    return 0;
}

